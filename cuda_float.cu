#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

bool InitCUDA(){
    int count;
    
    hipGetDeviceCount(&count);
    if(count == 0){
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(int i = 0; i<count;i++){
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess){
            if(prop.major >= 1){
                break;
            }
        }
    }
    
    if(i == count){
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

//产生矩阵
void matgen(float* a, int lda, int n){
    int i, j;

    for(i = 0; i < n; i++) {
        for(j = 0; j < n; j++) {
            a[i * lda + j] = (float) rand() / RAND_MAX + 
                (float) rand() / (RAND_MAX * RAND_MAX);
        }
    }
}
//矩阵乘法
void matmult(const float* a, int lda, const float* b, int ldb, 
    float* c, int ldc, int n){
    int i, j, k;

    for(i = 0; i < n; i++) {
        for(j = 0; j < n; j++) {
            double t = 0;
            for(k = 0; k < n; k++) {
                t += a[i * lda + k] * b[k * ldb + j];
            }
            c[i * ldc + j] = t;
        }
    }
}

//验证结果
void compare_mat(const float* a, int lda, 
    const float* b, int ldb, int n){
    float max_err = 0;
    float average_err = 0;
    int i, j;
    
    for(i = 0; i < n; i++) {
        for(j = 0; j < n; j++) {
            if(b[i * ldb + j] != 0) {
                float err = fabs((a[i * lda + j] -
                    b[i * ldb + j]) / b[i * ldb + j]);
                if(max_err < err) max_err = err;
                average_err += err;
            }
        }
    }

    printf("Max error: %g Average error: %g\n",
        max_err, average_err / (n * n));
}


//CUDA内计算1.0
__global__ static void matMultCUDA(const float* a, size_t lda,
    const float* b, size_t ldb, float* c, size_t ldc, int n){
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int idx = bid * blockDim.x + tid;
    const int row = idx / n;
    const int column = idx % n;
    int i;

    //改善误差前
    // if(row < n && column < n) {
    //     float t = 0;
    //     for(i = 0; i < n; i++) {
    //         t += a[row * lda + i] * b[i * ldb + column];
    //     }
    //     c[row * ldc + column] = t;
    // }
    //采用Kahan's Summation Formula改善误差后
    if(row < n && column < n) {
        float t = 0;
        float y = 0;
        for(i = 0; i < n; i++) {
            float r;
            y -= a[row * lda + i] * b[i * ldb + column];
            r = t - y;
            y = (r - t) + y;
            t = r;
        }
    }
}

//改良2.0
//  __global__ static void matMultCUDA(const float* a, size_t lda,
//     const float* b, size_t ldb, float* c, size_t ldc, int n){
//     extern __shared__ float data[];
//     const int tid = threadIdx.x;
//     const int row = blockIdx.x;
//     int i, j;

//     for(i = tid; i < n; i += blockDim.x) {
//         data[i] = a[row * lda + i];
//     }

//     __syncthreads();

//     for(j = tid; j < n; j += blockDim.x) {
//         float t = 0;
//         float y = 0;
//         for(i = 0; i < n; i++) {
//             float r;
//             y -= data[i] * b[i * ldb + j];
//             r = t - y;
//             y = (r - t) + y;
//             t = r;
//         }
//         c[row * ldc + j] = t;
//     }
// }

//改良3.0
//  __global__ static void matMultCUDA(const float* a, size_t lda,
//         const float* b, size_t ldb, float* c, size_t ldc, int n){
//     __shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
//     __shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
//     const int tidc = threadIdx.x;
//     const int tidr = threadIdx.y;
//     const int bidc = blockIdx.x * BLOCK_SIZE;
//     const int bidr = blockIdx.y * BLOCK_SIZE;
//     int i, j;

//     float results = 0;
//     float comp = 0;

//     for(j = 0; j < n; j += BLOCK_SIZE) {
//         if(tidr + bidr < n && tidc + j < n) {
//             matA[tidr][tidc] = a[(tidr + bidr) * lda + tidc + j];
//         }
//         else {
//             matA[tidr][tidc] = 0;
//         }

//         if(tidr + j < n && tidc + bidc < n) {
//             matB[tidr][tidc] = b[(tidr + j) * ldb + tidc + bidc];
//         }
//         else {
//             matB[tidr][tidc] = 0;
//         }

//         __syncthreads();

//         for(i = 0; i < BLOCK_SIZE; i++) {
//             float t;
//             comp -= matA[tidr][i] * matB[i][tidc];
//             t = results - comp;
//             comp = (t - results) + comp;
//             results = t;
//         }

//         __syncthreads();
//     }

//     if(tidr + bidr < n && tidc + bidc < n) {
//         c[(tidr + bidr) * ldc + tidc + bidc] = results;
//     }
// }

//改良4.0 
__global__ static void matMultCUDA(const float* a, size_t lda,
        const float* b, size_t ldb, float* c, size_t ldc, int n){
    __shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
    const int tidc = threadIdx.x;
    const int tidr = threadIdx.y;
    const int bidc = blockIdx.x * BLOCK_SIZE;
    const int bidr = blockIdx.y * BLOCK_SIZE;
    int i, j;

    float results = 0;
    float comp = 0;

    for(j = 0; j < n; j += BLOCK_SIZE) {
        matA[tidr][tidc] = a[(tidr + bidr) * lda + tidc + j];
        matB[tidr][tidc] = b[(tidr + j) * ldb + tidc + bidc];

        __syncthreads();

        for(i = 0; i < BLOCK_SIZE; i++) {
            float t;
            comp -= matA[tidr][i] * matB[i][tidc];
            t = results - comp;
            comp = (t - results) + comp;
            results = t;
        }

        __syncthreads();
    
    }
    
    c[(tidr + bidr) * ldc + tidc + bidc] = results;
}


//CUDA 矩阵乘法1.0
// #define NUM_THREADS 256

// clock_t matmultCUDA(const float* a, int lda,
//     const float* b, int ldb, float* c, int ldc, int n){
//     float *ac, *bc, *cc;
//     clock_t start, end;

//     start = clock();
//     hipMalloc((void**) &ac, sizeof(float) * n * n);
//     hipMalloc((void**) &bc, sizeof(float) * n * n);
//     hipMalloc((void**) &cc, sizeof(float) * n * n);

//     hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda,
//         sizeof(float) * n, n, hipMemcpyHostToDevice);
//     hipMemcpy2D(bc, sizeof(float) * n, b, sizeof(float) * ldb,
//         sizeof(float) * n, n, hipMemcpyHostToDevice);

//     int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    
//     //初始版本
//     // matMultCUDA<<<blocks * n, NUM_THREADS>>>
//     //     (ac, n, bc, n, cc, n, n);
    
//     //改良1.0
//     matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
//             (ac, n, bc, n, cc, n, n);

//     hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n,
//     sizeof(float) * n, n, hipMemcpyDeviceToHost);

//     hipFree(ac);
//     hipFree(bc);
//     hipFree(cc);

//     end = clock();

//     return end - start;
// }

//CUDA 矩阵乘法2.0
// #define NUM_THREADS 256

// clock_t matmultCUDA(const float* a, int lda,
//     const float* b, int ldb, float* c, int ldc, int n){
//     float *ac, *bc, *cc;
//     clock_t start, end;

//     start = clock();
//     // hipMalloc((void**) &ac, sizeof(float) * n * n);
//     // hipMalloc((void**) &bc, sizeof(float) * n * n);
//     // hipMalloc((void**) &cc, sizeof(float) * n * n);
//     //可以自动以最佳的倍数来配置记忆体
//     size_t pitch_a, pitch_b, pitch_c;
//     hipMallocPitch((void**) &ac, &pitch_a, sizeof(float) * n, n);
//     hipMallocPitch((void**) &bc, &pitch_b, sizeof(float) * n, n);
//     hipMallocPitch((void**) &cc, &pitch_c, sizeof(float) * n, n);

//     // hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda,
//     //     sizeof(float) * n, n, hipMemcpyHostToDevice);
//     // hipMemcpy2D(bc, sizeof(float) * n, b, sizeof(float) * ldb,
//     //     sizeof(float) * n, n, hipMemcpyHostToDevice);
//     //hipMallocPitch函数会以适当的倍数配置记忆体，并把配置的宽度传回
//     //因此，在把矩阵复制到显示记忆体上时，要使用它传回的宽度
//     hipMemcpy2D(ac, pitch_a, a, sizeof(float) * lda,
//         sizeof(float) * n, n, hipMemcpyHostToDevice);
//     hipMemcpy2D(bc, pitch_b, b, sizeof(float) * ldb,
//         sizeof(float) * n, n, hipMemcpyHostToDevice);


//     int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    
//     //初始版本
//     // matMultCUDA<<<blocks * n, NUM_THREADS>>>
//     //     (ac, n, bc, n, cc, n, n);
    
//     //改良1.0
//     // matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
//     //        (ac, n, bc, n, cc, n, n);


//     matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
//         (ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float),
//         cc, pitch_c / sizeof(float), n);
    
//     // hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n,
//     // sizeof(float) * n, n, hipMemcpyDeviceToHost);

//     hipMemcpy2D(c, sizeof(float) * ldc, cc, pitch_c,
//         sizeof(float) * n, n, hipMemcpyDeviceToHost);


//     hipFree(ac);
//     hipFree(bc);
//     hipFree(cc);

//     end = clock();

//     return end - start;
// }

//CUDA 矩阵乘法3.0 block
// #define NUM_THREADS 256

// clock_t matmultCUDA(const float* a, int lda,
//     const float* b, int ldb, float* c, int ldc, int n){
//     float *ac, *bc, *cc;
//     clock_t start, end;

//     start = clock();
//     // hipMalloc((void**) &ac, sizeof(float) * n * n);
//     // hipMalloc((void**) &bc, sizeof(float) * n * n);
//     // hipMalloc((void**) &cc, sizeof(float) * n * n);
//     //可以自动以最佳的倍数来配置记忆体
//     size_t pitch_a, pitch_b, pitch_c;
//     hipMallocPitch((void**) &ac, &pitch_a, sizeof(float) * n, n);
//     hipMallocPitch((void**) &bc, &pitch_b, sizeof(float) * n, n);
//     hipMallocPitch((void**) &cc, &pitch_c, sizeof(float) * n, n);

//     // hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda,
//     //     sizeof(float) * n, n, hipMemcpyHostToDevice);
//     // hipMemcpy2D(bc, sizeof(float) * n, b, sizeof(float) * ldb,
//     //     sizeof(float) * n, n, hipMemcpyHostToDevice);
//     //hipMallocPitch函数会以适当的倍数配置记忆体，并把配置的宽度传回
//     //因此，在把矩阵复制到显示记忆体上时，要使用它传回的宽度
//     hipMemcpy2D(ac, pitch_a, a, sizeof(float) * lda,
//         sizeof(float) * n, n, hipMemcpyHostToDevice);
//     hipMemcpy2D(bc, pitch_b, b, sizeof(float) * ldb,
//         sizeof(float) * n, n, hipMemcpyHostToDevice);


//     int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    
//     //初始版本
//     // matMultCUDA<<<blocks * n, NUM_THREADS>>>
//     //     (ac, n, bc, n, cc, n, n);
    
//     //改良1.0
//     // matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
//     //        (ac, n, bc, n, cc, n, n);

//     //改良2.0
//     // matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
//     //     (ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float),
//     //     cc, pitch_c / sizeof(float), n);

//     int bx = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
//     dim3 blocks(bx, bx);
//     dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//     matMultCUDA<<<blocks, threads>>>(ac, pitch_a / sizeof(float),
//         bc, pitch_b / sizeof(float), cc, pitch_c / sizeof(float), n);
    
//     // hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n,
//     // sizeof(float) * n, n, hipMemcpyDeviceToHost);

//     hipMemcpy2D(c, sizeof(float) * ldc, cc, pitch_c,
//         sizeof(float) * n, n, hipMemcpyDeviceToHost);


//     hipFree(ac);
//     hipFree(bc);
//     hipFree(cc);

//     end = clock();

//     return end - start;
// }

//CUDA 改良版4.0 配置好记忆体的倍数，同时清空为0
#define NUM_THREADS 256

clock_t matmultCUDA(const float* a, int lda,
    const float* b, int ldb, float* c, int ldc, int n){
    float *ac, *bc, *cc;
    clock_t start, end;

    start = clock();
    // hipMalloc((void**) &ac, sizeof(float) * n * n);
    // hipMalloc((void**) &bc, sizeof(float) * n * n);
    // hipMalloc((void**) &cc, sizeof(float) * n * n);
    //可以自动以最佳的倍数来配置记忆体
    size_t pitch_a, pitch_b, pitch_c;
    //改良版3.0
    // hipMallocPitch((void**) &ac, &pitch_a, sizeof(float) * n, n);
    // hipMallocPitch((void**) &bc, &pitch_b, sizeof(float) * n, n);
    // hipMallocPitch((void**) &cc, &pitch_c, sizeof(float) * n, n);
    int newn = ((n + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;

    hipMallocPitch((void**) &ac, &pitch_a,
        sizeof(float) * newn, newn);
    hipMallocPitch((void**) &bc, &pitch_b,
        sizeof(float) * newn, newn);
    hipMallocPitch((void**) &cc, &pitch_c,
        sizeof(float) * newn, newn);

    hipMemset(ac, 0, pitch_a * newn);
    hipMemset(bc, 0, pitch_b * newn);



    // hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda,
    //     sizeof(float) * n, n, hipMemcpyHostToDevice);
    // hipMemcpy2D(bc, sizeof(float) * n, b, sizeof(float) * ldb,
    //     sizeof(float) * n, n, hipMemcpyHostToDevice);
    //hipMallocPitch函数会以适当的倍数配置记忆体，并把配置的宽度传回
    //因此，在把矩阵复制到显示记忆体上时，要使用它传回的宽度
    hipMemcpy2D(ac, pitch_a, a, sizeof(float) * lda,
        sizeof(float) * n, n, hipMemcpyHostToDevice);
    hipMemcpy2D(bc, pitch_b, b, sizeof(float) * ldb,
        sizeof(float) * n, n, hipMemcpyHostToDevice);


    int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    
    //初始版本
    // matMultCUDA<<<blocks * n, NUM_THREADS>>>
    //     (ac, n, bc, n, cc, n, n);
    
    //改良1.0
    // matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
    //        (ac, n, bc, n, cc, n, n);

    //改良2.0
    // matMultCUDA<<<n, NUM_THREADS, sizeof(float) * n>>>
    //     (ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float),
    //     cc, pitch_c / sizeof(float), n);

    int bx = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(bx, bx);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    matMultCUDA<<<blocks, threads>>>(ac, pitch_a / sizeof(float),
        bc, pitch_b / sizeof(float), cc, pitch_c / sizeof(float), n);
    
    // hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n,
    // sizeof(float) * n, n, hipMemcpyDeviceToHost);

    hipMemcpy2D(c, sizeof(float) * ldc, cc, pitch_c,
        sizeof(float) * n, n, hipMemcpyDeviceToHost);


    hipFree(ac);
    hipFree(bc);
    hipFree(cc);

    end = clock();

    return end - start;
}

int main(){
    float *a, *b, *c, *d;
    int n = 1000;

    if(!InitCUDA()) return 0;

    a = (float*) malloc(sizeof(float) * n * n);
    b = (float*) malloc(sizeof(float) * n * n);
    c = (float*) malloc(sizeof(float) * n * n);
    d = (float*) malloc(sizeof(float) * n * n);

    srand(0);

    matgen(a, n, n);
    matgen(b, n, n);

    clock_t time = matmultCUDA(a, n, b, n, c, n, n);

    matmult(a, n, b, n, d, n, n);
    compare_mat(c, n, d, n, n);

    double sec = (double) time / CLOCKS_PER_SEC;
    printf("Time used: %.2f (%.2lf GFLOPS)\n", sec,
       2.0 * n * n * n / (sec * 1E9));

    return 0;    
}
