#include <stdio.h>
#include <hip/hip_runtime.h>

bool InitCUDA(){
    int count;
    
    hipGetDeviceCount(&count);
    if(count == 0){
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(int i = 0; i<count;i++){
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess){
            if(prop.major >= 1){
                break;
            }
        }
    }
    
    if(i == count){
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

int main(){
    if(!InitCUDA()){
        return 0;
    }
    
    printf("CUDA initialized.\n");

    return 0;
}
